#include "hip/hip_runtime.h"
#include "helpers.h"
#include <SDL2/SDL_events.h>
#include <cmath>
#include <cstdlib>
#include <cuda_device_runtime_api.h>
#include <hip/driver_types.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <time.h>


#define WIDTH 1920
#define HEIGHT 1080





__global__ void drawNodes(unsigned char* buffer, int width, int height, Node *nodes, int numNodes, float time, PlayerData* pData) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numNodes) return;

    Node n = nodes[idx];
    n.x += sinf(time*(10.0+ 50.0/n.value)/10.0f);
    n.y += cosf(time*(10.0+ 50.0/n.value)/10.0f);
    nodes[idx] = n;

    float radius = n.value*pData->zoom;
    float r2 = radius * radius;
    float aaWidth = 1.0f;  // anti-aliasing transition zone

    for (int u = -radius - aaWidth; u <= radius + aaWidth; u++) {
        for (int v = -radius - aaWidth; v <= radius + aaWidth; v++) {
            int x = (int)(n.x*pData->zoom + u + pData->px);
            int y = (int)(n.y*pData->zoom + v + pData->py);

            if (x < 0 || x >= width || y < 0 || y >= height) continue;

            float dist2 = u * u + v * v;
            float dist = sqrtf(dist2);
            float alpha = 0.0f;

            if (dist < radius - aaWidth) {
                alpha = 1.0f; // full opacity
            } else if (dist < radius + aaWidth) {
                // fade out
                alpha = (radius + aaWidth - dist) / (2.0f * aaWidth);
            } else {
                continue; // outside soft edge
            }

            int pi = (y * width + x) * 4;
            unsigned char a = (alpha * 255.0f);
            buffer[pi + 0] = (unsigned char)(255);
            buffer[pi + 1] = 0;
            buffer[pi + 2] = 0;
            buffer[pi + 3] = (unsigned char) a;
        }
    }
}

__device__ float lerp(float a, float b, float t) {
    return a + t * (b - a);
}



__device__ float clamp(float a, float b, float c){
    return min(max(a,b),c);
}
__global__ void drawEdges(unsigned char* buffer, int width, int height, Edge* edges, Node* nodes, int numEdges, PlayerData *pData)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numEdges) return;

    Edge e = edges[idx];
    Node n1 = nodes[e.n1];
    Node n2 = nodes[e.n2];

    int x0 = static_cast<int>(n1.x * pData->zoom + pData->px);
    int y0 = static_cast<int>(n1.y * pData->zoom + pData->py);
    int x1 = static_cast<int>(n2.x * pData->zoom + pData->px);
    int y1 = static_cast<int>(n2.y * pData->zoom + pData->py);

    int thickness = static_cast<int>((e.strength * 4.0f) + 2)*pData->zoom;

    int dx = abs(x1 - x0);
    int dy = abs(y1 - y0);
    int sx = (x0 < x1) ? 1 : -1;
    int sy = (y0 < y1) ? 1 : -1;
    int err = dx - dy;

    float dist = sqrtf((dx * dx) + (dy * dy));
    if (dist > 500.0f) return;
    while (true)
    {
        for (int ox = -thickness / 2; ox <= thickness / 2; ++ox)
        {
            for (int oy = -thickness / 2; oy <= thickness / 2; ++oy)
            {
                int px = x0 + ox;
                int py = y0 + oy;

                if (px >= 0 && px < width && py >= 0 && py < height)
                {
                    int i = (py * width + px) * 4;

                    buffer[i + 0] = 0;
                    buffer[i + 1] = 0;
                    buffer[i + 2] = (unsigned char) clamp((50000.0f / dist),0.0f,255.0f);
                    buffer[i + 3] = 0;
                }
            }
        }

        if (x0 == x1 && y0 == y1) break;

        int e2 = 2 * err;
        if (e2 > -dy)
        {
            err -= dy;
            x0 += sx;
        }
        if (e2 < dx)
        {
            err += dx;
            y0 += sy;
        }
    }
}



void checkCuda(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << msg << " (" << hipGetErrorString(err) << ")\n";
        exit(-1);
    }
}

bool initCUDA(unsigned char** d_buffer, int width, int height) {
    // allocate device buffer
    hipError_t err = hipMalloc(d_buffer, width * height * 4);
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: hipMalloc (" << hipGetErrorString(err) << ")\n";
        return false;
    }
    return true;
}

void cleanupCUDA(unsigned char* d_buffer) {
    if (d_buffer) {
        checkCuda(hipFree(d_buffer), "hipFree");
    }
}


int main() {
    // init SDL2 context
    SDL2Context sdlContext = {nullptr, nullptr, nullptr};
    if (!initSDL2(sdlContext,WIDTH,HEIGHT)) {
        return -1;
    }

    // Allocate device buffer for pixel data
    unsigned char* d_buffer = nullptr;
    checkCuda(hipMalloc(&d_buffer, WIDTH * HEIGHT * 4), "hipMalloc");

    // main loop
    bool running = true;
    bool mouseDown = false;
    SDL_Event event;
    int pixelX = WIDTH / 2;
    int pixelY = HEIGHT / 2;

    int numNodes;
    Node* h_nodes = readNodes("nodes.txt",&numNodes);
    Node* d_nodes;
    hipMalloc((void**)&d_nodes,sizeof(Node)*numNodes);
    hipMemcpy(d_nodes, h_nodes, sizeof(Node)*numNodes,hipMemcpyHostToDevice);


    int numEdges;
    Edge* h_edges = readEdges("edges.txt",&numEdges);
    Edge* d_edges;
    hipMalloc((void**)&d_edges,sizeof(Node)*numEdges);
    hipMemcpy(d_edges, h_edges, sizeof(Node)*numEdges,hipMemcpyHostToDevice);

    PlayerData pData = {0,0, 1.0}; 

    PlayerData *h_pData = &pData;
    PlayerData *d_pData;

    hipMalloc((void**) &d_pData,sizeof(PlayerData)); // maybe will have to move this logic to inside the game loop, so that d_pdata gets updated? or does it get set auto?
    hipMemcpy(d_pData,h_pData,sizeof(PlayerData),hipMemcpyHostToDevice);

    while (running) {
        // Handle events
        while (SDL_PollEvent(&event)) {
            if (event.type == SDL_MOUSEBUTTONDOWN)
            {
                mouseDown = true;
            }
            if (event.type == SDL_MOUSEBUTTONUP)
            {
                mouseDown = false;
            }
            if (event.type == SDL_MOUSEMOTION && mouseDown) {
                pData.px += event.motion.xrel;
                pData.py += event.motion.yrel;
            }
            if (event.type == SDL_MOUSEWHEEL) {
                float sensitivity = 0.1f;
                float oldZoom = pData.zoom;
                
                // Multiplicative zoom (better than additive)
                pData.zoom *= (1.0f + event.wheel.y * sensitivity);
                
                pData.zoom = fmax(0.1f, fmin(10.0f, pData.zoom));
                
                // Adjust pan to zoom from screen center
                pData.px = (WIDTH/2.0f) - ((WIDTH/2.0f - pData.px) * pData.zoom / oldZoom);
                pData.py = (HEIGHT/2.0f) - ((HEIGHT/2.0f - pData.py) * pData.zoom / oldZoom);
            }
            if (event.type == SDL_QUIT) {
                running = false;
            }

        }
        hipMemcpy(d_pData, &pData, sizeof(PlayerData), hipMemcpyHostToDevice); // updating player data after events
        // Clear device buffer
        checkCuda(hipMemset(d_buffer, 0, WIDTH * HEIGHT * 4), "hipMemset");

        float t = SDL_GetTicks() / 1000.0f;  // time in seconds as float
        // float t = 1.0;
        int blockSize = 256; // Good default
        int numBlocks = (numNodes + blockSize - 1) / blockSize;

        

        numBlocks = (numEdges + blockSize - 1 ) / blockSize;
        drawEdges<<<numBlocks,blockSize>>>(d_buffer,WIDTH,HEIGHT,d_edges,d_nodes,numEdges,d_pData);

        checkCuda(hipGetLastError(), "kernel launch");
        checkCuda(hipDeviceSynchronize(),"kernel sync");
        
        drawNodes<<<numBlocks, blockSize>>>(d_buffer, WIDTH, HEIGHT, d_nodes,numNodes,t ,d_pData);

        checkCuda(hipGetLastError(), "kernel launch");
        checkCuda(hipDeviceSynchronize(), "kernel sync");
        // lock texture and copy CUDA buffer to texture
        void* pixels = nullptr;
        int pitch;
        if (SDL_LockTexture(sdlContext.texture, nullptr, &pixels, &pitch) != 0) {
            std::cerr << "SDL_LockTexture failed: " << SDL_GetError() << "\n";
            running = false;
        } else {
            checkCuda(hipMemcpy(pixels, d_buffer, WIDTH * HEIGHT * 4, hipMemcpyDeviceToHost), "hipMemcpy to texture");
            SDL_UnlockTexture(sdlContext.texture);
        }

        // sdl render
        SDL_RenderClear(sdlContext.renderer);
        SDL_RenderCopy(sdlContext.renderer, sdlContext.texture, nullptr, nullptr);
        SDL_RenderPresent(sdlContext.renderer);
    }

    // cleanup
    checkCuda(hipFree(d_buffer), "hipFree");
    checkCuda(hipFree(d_edges),"hipFree");
    delete[] h_nodes;
    delete[] h_edges;
    cleanupSDL2(sdlContext);
    return 0;
}